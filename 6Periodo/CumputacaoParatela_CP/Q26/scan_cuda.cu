
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


/* ===================================     scan_cuda.cu       ===================================

a[39999999] = 799999980000000.000000

real    0m2.485s
user    0m1.233s
sys     0m1.130s

==27669== NVPROF is profiling process 27669, command: ./scan_cuda

a[39999999] = 799999980000000.000000
==27669== Profiling application: ./scan_cuda
==27669== Profiling result:
Time(%)      Time     Calls       Avg       Min       Max  Name
 51.51%  464.14ms         2  232.07ms  452.72us  463.69ms  [CUDA memcpy HtoD]
 41.72%  375.87ms         2  187.94ms  362.41us  375.51ms  [CUDA memcpy DtoH]
  5.10%  45.915ms         1  45.915ms  45.915ms  45.915ms  scan_cuda(double*, double*, int)
  1.68%  15.118ms         1  15.118ms  15.118ms  15.118ms  add_cuda(double*, double*, int)

==27669== API calls:
Time(%)      Time     Calls       Avg       Min       Max  Name
 77.63%  902.05ms         4  225.51ms  94.699us  463.95ms  cudaMemcpy
 22.23%  258.28ms         2  129.14ms  8.6740us  258.27ms  cudaMalloc
  0.06%  717.03us         2  358.52us  40.549us  676.49us  cudaFree
  0.04%  483.83us        90  5.3750us     292ns  205.79us  cuDeviceGetAttribute
  0.02%  255.32us         2  127.66us  33.083us  222.24us  cudaLaunch
  0.01%  92.437us         1  92.437us  92.437us  92.437us  cuDeviceTotalMem
  0.01%  65.884us         1  65.884us  65.884us  65.884us  cuDeviceGetName
  0.00%  13.031us         6  2.1710us     360ns  9.7500us  cudaSetupArgument
  0.00%  6.7530us         2  3.3760us  1.4020us  5.3510us  cudaConfigureCall
  0.00%  2.7420us         2  1.3710us     990ns  1.7520us  cuDeviceGetCount
  0.00%  1.1170us         2     558ns     499ns     618ns  cuDeviceGet
  
===================================     scan.c    ===================================
a[39999999] = 799999980000000.000000

real    0m0.511s
user    0m0.216s
sys     0m0.287s

*/

__global__ void scan_cuda(double* a, double *s, int width) {
  int t = threadIdx.x;
  int b = blockIdx.x*blockDim.x;
  double fodase;

  __shared__ double p[1024];
	
  if(b + t < width) {
    p[t] = a[t+b];
  }
  __syncthreads();

  for(int i = 1; i < blockDim.x; i = i * 2) {
    if (t >= i){
	  fodase = p[t] + p[t-i];
	}
	__syncthreads();
	if (t >= i){
      p[t] = fodase;
	}
	__syncthreads();
  }
  if (b + t < width){
    a[t+b] = p[t];
  }
  if (t == blockDim.x-1){
    s[blockIdx.x+1] = a[t+b];
  }
} 

__global__ void add_cuda(double *a, double *s, int width) {
  int t = threadIdx.x;
  int b = blockIdx.x*blockDim.x;

  if (b + t < width) {
    a[b+t] += s[blockIdx.x];
  }
}

int main()
{
  int width = 40000000;
  int size = width * sizeof(double);

  int block_size = 1024;
  int num_blocks = (width-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));  
 
  double *a = (double*) malloc (size);
  double *s = (double*) malloc (s_size);

  for(int i = 0; i < width; i++)
    a[i] = i;

  double *d_a, *d_s;
  
  // alocar vetores "a" e "s" no device
	hipMalloc((void**) &d_a, size);
	hipMalloc((void**) &d_s, s_size);

  // copiar vetor "a" para o device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

  // definição do número de blocos e threads (dimGrid e dimBlock)
	dim3 dimGrid(num_blocks, 1, 1);
	dim3 dimBlock(block_size, 1, 1);

  // chamada do kernel scan
	scan_cuda <<<dimGrid,dimBlock>>>(d_a, d_s, width);
  
  // copiar vetor "s" para o host
	hipMemcpy(s, d_s, s_size, hipMemcpyDeviceToHost); 

  // scan no host (já implementado)
  s[0] = 0;
  for (int i = 1; i < num_blocks; i++)
    s[i] += s[i-1];
 
  // copiar vetor "s" para o device
	hipMemcpy(d_s, s, s_size, hipMemcpyHostToDevice);

  // chamada do kernel da soma
	add_cuda <<<dimGrid,dimBlock>>>(d_a, d_s, width);
  
  // copiar o vetor "a" para o host
	hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);

  printf("\na[%d] = %f\n",width-1,a[width-1]);
  
  hipFree(d_a);
  hipFree(d_s);
}